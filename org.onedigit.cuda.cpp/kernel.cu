
#include "hip/hip_runtime.h"


#include <iostream>
#include <stdio.h>
#include <stdlib.h>

#define N 10

__global__ void add(int a, int b, int *c)
{
	*c = a + b;
}

__global__ void vecAddDevice(int* a, int* b, int* c)
{
	int gx = gridDim.x;
	int gy = gridDim.y;
	int tid = blockIdx.x; 
	/* printf("tid = %d\n", tid); */
	if (tid < N) {
		c[tid] = a[tid] + b[tid];
	}
}

void getDevice()
{
	int count;
	int error =	hipGetDeviceCount(&count);
    if (error != hipSuccess) {
        printf("hipGetDeviceCount returned error code %d, line(%d)\n", error, __LINE__);
        exit(EXIT_FAILURE);
    }
    std::cout << "Number of devices = " << count << std::endl;
    hipDeviceProp_t prop;
    for (int i = 0; i < count; i++) {
		error = hipGetDeviceProperties(&prop, i);
		if (error == hipSuccess) {
			std::cout << "Name: " << prop.name << std::endl;
			std::cout << "Total Global Mem: " << prop.totalGlobalMem << std::endl;
		}
	}
}

void vecAdd()
{
	getDevice();

	int a[N], b[N], c[N];
	int *dev_a, *dev_b, *dev_c;
	int error = hipMalloc( (void**)&dev_a, N * sizeof(int) );
    if (error != hipSuccess) {
        printf("hipMalloc returned error code %d, line(%d)\n", error, __LINE__);
        exit(EXIT_FAILURE);
    }
	error = hipMalloc( (void**)&dev_b, N * sizeof(int) );
    if (error != hipSuccess) {
        printf("hipMalloc returned error code %d, line(%d)\n", error, __LINE__);
        exit(EXIT_FAILURE);
    }
	error = hipMalloc( (void**)&dev_c, N * sizeof(int) );
    if (error != hipSuccess) {
        printf("hipMalloc returned error code %d, line(%d)\n", error, __LINE__);
        exit(EXIT_FAILURE);
    }
	for (int i = 0; i < N; i++) {
		a[i] = -i;
		b[i] = i * i;
	}
	error = hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice);
    if (error != hipSuccess) {
        printf("hipMemcpy returned error code %d, line(%d)\n", error, __LINE__);
        exit(EXIT_FAILURE);
    }
	error = hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice);
    if (error != hipSuccess) {
        printf("hipMemcpy returned error code %d, line(%d)\n", error, __LINE__);
        exit(EXIT_FAILURE);
    }

	vecAddDevice<<<N, 1>>>(dev_a, dev_b, dev_c);
	
	error = hipMemcpy(c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost);
    if (error != hipSuccess) {
        printf("hipMemcpy returned error code %d, line(%d)\n", error, __LINE__);
        exit(EXIT_FAILURE);
    }
	/*
	for (int i = 0; i < N; i++) {
		std::cout << a[i] << "+" << b[i] << " = " << c[i] << std::endl;
	}
	*/
}

void test()
{
	getDevice();

	int c;
	int* dev_c;
	int error = hipMalloc( (void**)&dev_c, sizeof(int) );
    if (error != hipSuccess) {
        printf("hipMalloc returned error code %d, line(%d)\n", error, __LINE__);
        exit(EXIT_FAILURE);
    }
	
	add<<<1, 1>>>(2, 7, dev_c);
	
	error =	hipMemcpy(&c, dev_c, sizeof(int), hipMemcpyDeviceToHost);
    if (error != hipSuccess) {
        printf("hipMemcpy returned error code %d, line(%d)\n", error, __LINE__);
        exit(EXIT_FAILURE);
    }

	std::cout << "2 + 7 = " << c << std::endl;
	hipFree(dev_c);
}

int main()
{
	vecAdd();
	// test();
	return 0;
}

